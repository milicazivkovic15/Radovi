/* Problem - Naci najveci poligon od ucitanih tacaka */
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

#define TPB 16

__global__ void calculate(double *x, double *y, double *z, double *out, int n){
		int index=threadIdx.x+blockIdx.x*blockDim.x;
		__shared__ double temp[TPB];
		
		
		if (index+2<n){
			temp[threadIdx.x]=sqrt((x[index]-x[0]) * (x[index]-x[0])+ (y[index]-y[0])* (y[index]-y[0])+ (z[index]-z[0])* (z[index]-z[0]));
		}
	__syncthreads();
	if (threadIdx.x==0){
		double s=temp[0];
		int i ;
		for(i=1;i<TPB;i++){
			if (temp[i]>s) s=temp[i];
		}
		out[blockIdx.x]=s;
	}
}

int main(int argc, char **argv){
	
	double *x,*y,*z,*out,*d_x,*d_y,*d_z,*d_out;
	
	
	FILE *f=fopen("tacke.txt","r");
	int n;
	fscanf(f,"%d",&n);
	n++;	
	
	int size=sizeof(double)*n, outSize=sizeof(double)*(n-2+TPB-1)/TPB,i;

	hipMalloc((void**)&d_x,size);
	hipMalloc((void**)&d_y,size);
	hipMalloc((void**)&d_z,size);
	hipMalloc((void**)&d_out,outSize);

	x=(double*)malloc(size);
	y=(double*)malloc(size);
	z=(double*)malloc(size);
	out=(double*)malloc(outSize);
	
	fscanf(f,"%lf%lf%lf",&(x[0]),&(y[0]),&(z[0]));
	double minx=x[1],miny=y[1],minz=z[1];
	double maxx=x[1],maxy=y[1],maxz=z[1];

	for(i=2;i<n;i++){
		
		fscanf(f,"%lf%lf%lf",&(x[i]),&(y[i]),&(z[i]));
		x[0]+=x[i];
		if (x[i]<minx) minx=x[i];
		else maxx=x[i];
		y[0]+=y[i];
		if (y[i]<miny) miny=y[i];
		else maxy=y[i];
		z[0]+=z[i];
		if (z[i]<minz) minz=z[i];
		else maxz=z[i];
	}
	x[0]/=(n-1);
	y[0]/=(n-1);
	z[0]/=(n-1);

	

	hipMemcpy(d_x,x,sizeof(double*)*n,hipMemcpyHostToDevice);
	hipMemcpy(d_y,y,sizeof(double*)*n,hipMemcpyHostToDevice);
	hipMemcpy(d_z,z,sizeof(double*)*n,hipMemcpyHostToDevice);

	calculate<<<(n-2+TPB-1)/TPB,TPB>>>(d_x,d_y,d_z,d_out,n);
	
	hipMemcpy(out,d_out,outSize,hipMemcpyDeviceToHost);
	
	double max=-9999;
	for (i=0;i<(n-2+TPB-1)/TPB;i++){
		if (max<out[i]) max=out[i];
	}
	printf("Precnik sfere je : %lf \n x=%lf y=%lf z=%lf\n",max,x[0],y[0],z[0]);
	fclose(f);
	return 0;
}
